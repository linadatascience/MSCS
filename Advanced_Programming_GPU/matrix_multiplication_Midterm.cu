#include "hip/hip_runtime.h"
#include "C:/Users/lmi/Documents/Visual Studio 2010/Projects/matrix_multiplication/common/book.h"
#include<hip/hip_runtime.h>

#define N 128
//const int threadPerBlock=256;
//const int blockPerGrid=imin(32,(N+threadPerBlock-1)/threadPerBlock);


__global__ void multiplication(float *a, float *b, float *c, int M) {
	int col = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	float sum=0;
	if(col<M && row<M){ 
		for(int k=0; k<M; k++)
			sum+=a[row*M+k]*b[k*M+col];
//		col+=grid.x*gridDim.x;
//		row+=grid.y*gridDim.y;
	    c[row*M+col]=sum;	
	}
}

int main(){
	float a[N][N], b[N][N],c[N][N],c_test[N][N];
	float *dev_a, *dev_b, *dev_c;
	dim3 blocks(N/16,N/16);
	dim3 threads(16,16);

	
	HANDLE_ERROR(hipMalloc((void **)&dev_a, N*N*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void **)&dev_b, N*N*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void **)&dev_c, N*N*sizeof(float)));

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			a[i][j]=i;
			b[i][j]=2*i;
		}
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice));

	multiplication<<<blocks, threads>>>(dev_a, dev_b, dev_c, N);

	HANDLE_ERROR(hipMemcpy(c, dev_c, N*N*sizeof(float), hipMemcpyDeviceToHost));

	
	for(int i=0; i<N; i++){
		for(int j=0; j<N;j++){
			float tem=0;
			for(int k=0;k<N;k++)
				tem+=a[i][k]+b[k][j];
			c_test[i][j]=tem;
		}
	}

	bool success=true;
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){
			if(c[i][j]!=c_test[i][j]) success=false;
		}
	}

	if(success) printf("we made it!");

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	delete [] a;
	delete [] b;
	delete [] c;
	delete [] c_test;

	return (0);
}