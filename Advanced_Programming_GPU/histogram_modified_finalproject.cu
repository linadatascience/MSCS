/* TO DO: Please put your name and date of modification
 *
 * Author:      Brady Chen  5/1/2015
 * Modified By:
 *              <your name> <date>    
 *
 * This is a C code for the computation of a histogram of data from an input text file. The
 * text file contains multiple lines of characters. The code generate the frequency histogram 
 * of characters from the input file.   
 */
                                                                                                    

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

/* TO DO: (possible)
 *
 * Definition of important values:
 *   MAX_TEXT_LINES  -- maximum number of lines for the input file. We initially set to 1000
 *                     If your file contains more than 1000, just change the value
 *   MAX_LINE_LENGTH -- the length of each line. You change the value
 *   NUMBER_OF_CHARS -- this the total number of characters used in the input file. We set 
 *                     the number to 128 which could include all the standard characters in
 *                     the ASCII code table. If your input file contains more than 128 standard
 *                     characters, please increase the number. For example, you should use 256
 *                     for extended ASCII code.
 *   TOTAL_SIZE      -- this is the total number of characters in the input file.
 * 
 * NOTE: You can change the values of the varibales if necessary but please don't change the variable
 *       names themselves  
 */ 
#define MAX_TEXT_LINES 4400  
#define MAX_LINE_LENGTH 75   
#define NUMBER_OF_CHARS 128   
#define TOTAL_SIZE (MAX_TEXT_LINES + 10)*(MAX_LINE_LENGTH + 10)

/*
 * Declaraion of the arrays of characters 
 *   aTextData    -- this is a two dimensional array of char. It contains all the characters 
 *                   inputted from the input file. The first index record the lines and the
 *                   second index record the columns in each line. We add 10 on MAX_TEXT_LINES 
 *                   and MAX_LINE_LENGTH as safety spaces to crash on off-by-one errors. 
 *
 *   NOTE: No need to change this part
 */
char aTextData[MAX_TEXT_LINES + 10][MAX_LINE_LENGTH + 10];
char aFlattenedData[(MAX_TEXT_LINES + 10)*(MAX_LINE_LENGTH + 10)];


/* TO DO:
 *
 * Declaration of function
 *   histogram() -- This function takes a two arguments
 *                  aText -- a two dimensional array of char.
 *                  histo -- a one dimensional array which returns the frequency histogram
 *                           of the characrers in the array aText
 * NOTE: you need to modify the declararion accordingly based on the changes in the gpu_histogram()
 */
int histogram(char aText[MAX_TEXT_LINES + 10][MAX_LINE_LENGTH + 10], unsigned int histo[NUMBER_OF_CHARS]);

int main(void) 
{
	/*
	 *  Declaration of variables 
	 *    histo      -- contains the frequency histogram of the characters
	 *    lineBuffer -- line buffer for reading
	 *    iTextSize  -- how large is the array
	 *    iTextUsed  -- how many lines contained
	 */
	unsigned int histo[NUMBER_OF_CHARS];
	char lineBuffer[MAX_LINE_LENGTH + 10];
	int  iTextSize = MAX_TEXT_LINES;  
	int  iTextUsed = 0;               

	// clean the memory with zeros
	memset(aTextData, 0, sizeof(aTextData));
	memset(lineBuffer, 0, sizeof(lineBuffer));

	/*  TO DO: (possible)
	 *  Data read and conversion: The following lines of code do the following
	 *    1. Find the input file (in this case "input.txt)
	 *    2. Read the data from the input file and store it in a two dimensional
	 *       array of char
	 * 
	 *  NOTE: Please DO NOT change the following lines of code except for the first
	 *        line, where you are allowed to change the path for the input file
	 *        "input.txt"
	 */
	char *inputFile = "../input_test.txt"; // you may need to change the path depending on
	                                  // where you put the input file
	// read input file into the array. use binary mode "rb"
	FILE *fin = fopen(inputFile, "rb");
	if (!fin) {
		printf("cannot read %s\n", inputFile);
		return 1;
	}
	// while not end of file, read another line
	while (fgets(lineBuffer, sizeof(lineBuffer)-10, fin) != 0)
	{
		// check if array has space for another line
		if (iTextUsed >= iTextSize)
		{
			printf("overflow: too many text lines\n"); 
			return 1;
		}
		// strip CR/LF from line endings so we get pure text
		char *psz = strchr(lineBuffer, '\r'); if (psz) *psz = '\0';
		psz = strchr(lineBuffer, '\n'); if (psz) *psz = '\0';
		// store the line
		strncpy(aTextData[iTextUsed], lineBuffer, MAX_LINE_LENGTH);
		aTextData[iTextUsed][MAX_LINE_LENGTH - 1] = '\0'; // safety
		iTextUsed++;
	}
	fclose(fin);
	/* end of the Data read and conversion */

	/* TO DO:
	 * Function call to compute the frequency histogram of the data
	 *   NOTE: you NEED TO MODIFY this portion.
	 *         1. declare the blocks and threads
	 *         2. call the newly implemented kernel function gpu_histogram() .
	 *         Please note that the kernel function DOES NOT take array as argument,
	 *         not to mention two dimensional array in this case.
	 */
	if (histogram(aTextData, histo) !=0) 
		return 1;
	
	/* TO DO:
	 *
	 * Output of histogram results
	 * the following is the output of the histogram frequency of characters in
	 * the order of ASCII code. the array histo[] contains the frequency histogram
	 * of 128 characters defined in ASCII.
	 *   1. The decimal values for the letters are
	 *      A to Z -- 65 to 90
	 *      a to z -- 97 to 122
	 *   2. the following lines of code print the histogram results for a to z.
	 *   
	 * NOTE: You are required to add code for computing the frequency histogram for
	 *       letters A to Z (including both low case and up case). To do this, you 
	 *       could add the frequency histograms for capital letters and low case letters.
	 *       For example, to calcuate the frequency histogram of letter A, you should
	 *       add the frequency histogram of capital A and low case a.  
	 */
	unsigned int histocount = 0; // total character count
	printf("histogram frequency of characters a to z: \n");
	for (int i = 97; i < 97+26 ; i++) {
		histocount += histo[i];
		printf("%d ", histo[i]);
	}
	printf("\n");
	printf("histogram frequency of characters in terms of percentiles: \n");
	for (int i = 97; i < 97 + 26; i++) {
		printf("%.4f ", (float)histo[i]/(float)histocount);
	}

	/* TO DO: Please add your code here for computing the frequency histogram for all letters
	 *        regardless low or up cases.
	 *
	 */

	return 0;
}

/* TO DO:
 *
 * Implementation of function
 *   histogram() -- This function takes a two arguments
 *                  aText -- a two dimensional array of char.
 *                  histo -- a one dimensional array which returns the frequency histogram
 *                           of the characrers in the array aText
 * NOTE: you need to change the function to kenel function. Here are the changes you need to make
 *        1. change the name of the function to gpu_histogram()
 *        2. change the arguments to piniters 
 *           gpu_histogram(char * aText, unsigned int * histo)
 *        3. re-implement the function
 */
int histogram(char aText[MAX_TEXT_LINES + 10][MAX_LINE_LENGTH + 10], unsigned int histo[NUMBER_OF_CHARS])
{
	int i, j;
	for (i = 0; i < NUMBER_OF_CHARS; i++)
		histo[i] = 0;
	for (i = 0; i < MAX_TEXT_LINES+10; i++) {
		for (j = 0; j < MAX_LINE_LENGTH + 10; j++) {
			histo[(int)(aText[i][j])]++;
		}
	}
	return 0;
}
