#include "hip/hip_runtime.h"
#include <C:/Program Files (x86)/Microsoft Visual Studio 14.0/VC/include/common/book.h>

#define N 64

__global__ void add(int *a, int *b, int *c) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	while (x < N & y < N) 
		c[y][x] = a[y][x] + b[y][x];
	    x+=grid.x*gridDim.x;
		y+=grid.y*gridDim.y;
}

int main(void) {
	int a[N][N], b[N][N], c[N][N];
	int (*dev_a)[N][N], (*dev_b)[N][N], (*dev_c)[N][N];
	dim3 blocks(N/16,N/16);
	dim3 threads(16,16);

	//allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**) &dev_a, N*N*sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, N*N*sizeof(int));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, N*N*sizeof(int));

	//initiate the matrixes that will be added
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			a[i][j] = i*N + j;
			b[i][j] = 2 * (i*N + j);

		}
	}

	//copy initialized matrixes a and b to GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N*N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*N*sizeof(int), hipMemcpyHostToDevice));

	//launch the addition function on GPU
	add << <blocks, threads >> > (dev_a, dev_b, dev_c);

	//copy the sum of two matrixes back to CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost));

	//display the result of addition of two matrixes
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			print("%d+%d=%d\n", a[i][j], b[i][j], c[i][j]);
		}
	}
}